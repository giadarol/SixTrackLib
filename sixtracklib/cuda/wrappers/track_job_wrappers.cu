#include "sixtracklib/cuda/wrappers/track_job_wrappers.h"
#include "sixtracklib/cuda/control/kernel_config.h"

#include <hip/hip_runtime_api.h>

#include "sixtracklib/common/definitions.h"
#include "sixtracklib/common/buffer/buffer_type.h"
#include "sixtracklib/common/control/definitions.h"
#include "sixtracklib/common/control/argument_base.h"
#include "sixtracklib/common/particles/definitions.h"
#include "sixtracklib/common/track/definitions.h"

#include "sixtracklib/cuda/definitions.h"
#include "sixtracklib/cuda/argument.h"
#include "sixtracklib/cuda/control/kernel_config.h"

#include "sixtracklib/cuda/kernels/extract_particles_addr.cuh"
#include "sixtracklib/cuda/kernels/be_monitors_assign_out_buffer.cuh"
#include "sixtracklib/cuda/kernels/elem_by_elem_assign_out_buffer.cuh"
#include "sixtracklib/cuda/kernels/assign_address_item.cuh"
#include "sixtracklib/cuda/kernels/track_particles.cuh"

void NS(Track_particles_until_turn_cuda_wrapper)(
    const NS(CudaKernelConfig) *const SIXTRL_RESTRICT kernel_config,
    NS(CudaArgument)* SIXTRL_RESTRICT particles_arg,
    NS(buffer_size_t) const pset_index,
    NS(CudaArgument)* SIXTRL_RESTRICT beam_elem_arg,
    NS(buffer_size_t) const until_turn,
    NS(CudaArgument)* SIXTRL_RESTRICT dbg_register_arg )
{
    dim3 const* ptr_blocks =
        NS(CudaKernelConfig_get_ptr_const_blocks)( kernel_config );

    bool const is_finished = NS(KernelConfig_needs_update)( kernel_config );

    dim3 const* ptr_threads =
        NS(CudaKernelConfig_get_ptr_const_threads_per_block)( kernel_config );

    if( dbg_register_arg == SIXTRL_NULLPTR )
    {
        SIXTRL_ASSERT( ptr_blocks  != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( ptr_threads != SIXTRL_NULLPTR );

        SIXTRL_ASSERT( !NS(KernelConfig_needs_update)( kernel_config ) );
        SIXTRL_ASSERT( NS(KernelConfig_get_arch_id)( kernel_config ) ==
            NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( particles_arg != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( NS(Argument_get_arch_id)( particles_arg ) ==
                   NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( particles_arg ) );
        SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
            particles_arg ) != SIXTRL_NULLPTR );

        SIXTRL_ASSERT( beam_elem_arg != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( NS(Argument_get_arch_id)( beam_elem_arg ) ==
                   NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( beam_elem_arg ) );
        SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
            beam_elem_arg ) != SIXTRL_NULLPTR );

        SIXTRL_ASSERT(
            NS(Argument_get_cobjects_buffer_slot_size)( particles_arg ) ==
            NS(Argument_get_cobjects_buffer_slot_size)( beam_elem_arg ) );

        NS(Track_particles_until_turn_cuda)<<< *ptr_blocks, *ptr_threads >>>(
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                particles_arg ), pset_index,
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                beam_elem_arg ), until_turn,
            NS(Argument_get_cobjects_buffer_slot_size)( particles_arg ) );
    }
    else if( ( ptr_blocks != SIXTRL_NULLPTR ) &&
        ( ptr_threads != SIXTRL_NULLPTR ) &&
        ( NS(KernelConfig_get_arch_id)( kernel_config ) ==
                NS(ARCHITECTURE_CUDA) ) &&
        ( !NS(KernelConfig_needs_update)( kernel_config ) ) )
    {
        NS(Track_particles_until_turn_cuda_debug)<<< *ptr_blocks, *ptr_threads >>>(
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                particles_arg ), pset_index,
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                beam_elem_arg ), until_turn,
            NS(Argument_get_cobjects_buffer_slot_size)( particles_arg ),
            NS(CudaArgument_get_cuda_arg_buffer_as_debugging_register_begin)(
                dbg_register_arg ) );
    }
}

void NS(Track_particles_elem_by_elem_until_turn_cuda_wrapper)(
    const NS(CudaKernelConfig) *const SIXTRL_RESTRICT kernel_config,
    NS(CudaArgument)* SIXTRL_RESTRICT particles_arg,
    NS(buffer_size_t) const pset_index,
    NS(CudaArgument)* SIXTRL_RESTRICT beam_elem_arg,
    NS(CudaArgument)* SIXTRL_RESTRICT config_buffer_arg,
    NS(buffer_size_t) const elem_by_elem_config_index,
    NS(buffer_size_t) const until_turn_elem_by_elem,
    NS(CudaArgument)* SIXTRL_RESTRICT dbg_register_arg )
{
    dim3 const* ptr_blocks = NS(CudaKernelConfig_get_ptr_const_blocks)(
        kernel_config );

    dim3 const* ptr_threads =
        NS(CudaKernelConfig_get_ptr_const_threads_per_block)( kernel_config );

    if( dbg_register_arg == SIXTRL_NULLPTR )
    {
        SIXTRL_ASSERT( ptr_blocks  != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( ptr_threads != SIXTRL_NULLPTR );

        SIXTRL_ASSERT( !NS(KernelConfig_needs_update)( kernel_config ) );
        SIXTRL_ASSERT( NS(KernelConfig_get_arch_id)( kernel_config ) ==
            NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( particles_arg != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( NS(Argument_get_arch_id)( particles_arg ) ==
                   NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( particles_arg ) );
        SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
            particles_arg ) != SIXTRL_NULLPTR );

        SIXTRL_ASSERT( beam_elem_arg != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( NS(Argument_get_arch_id)( beam_elem_arg ) ==
                   NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( beam_elem_arg ) );
        SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
            beam_elem_arg ) != SIXTRL_NULLPTR );

        SIXTRL_ASSERT( config_buffer_arg != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( NS(Argument_get_arch_id)( config_buffer_arg ) ==
                   NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( config_buffer_arg ) );
        SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
            config_buffer_arg ) != SIXTRL_NULLPTR );

        SIXTRL_ASSERT(
            NS(Argument_get_cobjects_buffer_slot_size)( particles_arg ) ==
            NS(Argument_get_cobjects_buffer_slot_size)( beam_elem_arg ) );

        SIXTRL_ASSERT(
            NS(Argument_get_cobjects_buffer_slot_size)( config_buffer_arg ) ==
            NS(Argument_get_cobjects_buffer_slot_size)( particles_arg ) );

        NS(Track_track_elem_by_elem_until_turn_cuda)<<< *ptr_blocks, *ptr_threads >>>(
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                particles_arg ), pset_index,
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                beam_elem_arg ),
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                config_buffer_arg ), elem_by_elem_config_index,
            until_turn_elem_by_elem,
            NS(Argument_get_cobjects_buffer_slot_size)( particles_arg ) );
    }
    else if( ( ptr_blocks != SIXTRL_NULLPTR ) &&
        ( ptr_threads != SIXTRL_NULLPTR ) &&
        ( NS(KernelConfig_get_arch_id)( kernel_config ) ==
                NS(ARCHITECTURE_CUDA) ) &&
        ( !NS(KernelConfig_needs_update)( kernel_config ) ) )
    {
        NS(Track_track_elem_by_elem_until_turn_cuda_debug)<<< *ptr_blocks, *ptr_threads >>>(
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                particles_arg ), pset_index,
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                beam_elem_arg ),
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                config_buffer_arg ), elem_by_elem_config_index,
            until_turn_elem_by_elem,
            NS(Argument_get_cobjects_buffer_slot_size)( particles_arg ),
            NS(CudaArgument_get_cuda_arg_buffer_as_debugging_register_begin)(
                dbg_register_arg ) );
    }
}

void NS(Track_particles_line_cuda_wrapper)(
    const NS(CudaKernelConfig) *const SIXTRL_RESTRICT kernel_config,
    NS(CudaArgument)* SIXTRL_RESTRICT particles_arg,
    NS(buffer_size_t) const pset_index,
    NS(CudaArgument)* SIXTRL_RESTRICT beam_elem_arg,
    NS(buffer_size_t) const be_begin_idx,
    NS(buffer_size_t) const be_end_idx,
    bool const finish_turn,
    NS(CudaArgument)* SIXTRL_RESTRICT dbg_register_arg )
{
    dim3 const* ptr_blocks = NS(CudaKernelConfig_get_ptr_const_blocks)(
        kernel_config );

    dim3 const* ptr_threads =
        NS(CudaKernelConfig_get_ptr_const_threads_per_block)( kernel_config );

    if( dbg_register_arg == SIXTRL_NULLPTR )
    {
        SIXTRL_ASSERT( ptr_blocks  != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( ptr_threads != SIXTRL_NULLPTR );

        SIXTRL_ASSERT( !NS(KernelConfig_needs_update)( kernel_config ) );
        SIXTRL_ASSERT( NS(KernelConfig_get_arch_id)( kernel_config ) ==
            NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( particles_arg != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( NS(Argument_get_arch_id)( particles_arg ) ==
                   NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( particles_arg ) );
        SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
            particles_arg ) != SIXTRL_NULLPTR );

        SIXTRL_ASSERT( beam_elem_arg != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( NS(Argument_get_arch_id)( beam_elem_arg ) ==
                   NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( beam_elem_arg ) );
        SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
            beam_elem_arg ) != SIXTRL_NULLPTR );

        SIXTRL_ASSERT(
            NS(Argument_get_cobjects_buffer_slot_size)( particles_arg ) ==
            NS(Argument_get_cobjects_buffer_slot_size)( beam_elem_arg ) );

        NS(Track_particles_line_cuda)<<< *ptr_blocks, *ptr_threads >>>(
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                particles_arg ), pset_index,
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                beam_elem_arg ),
                be_begin_idx, be_end_idx, finish_turn,
            NS(Argument_get_cobjects_buffer_slot_size)( particles_arg ) );
    }
    else if( ( ptr_blocks != SIXTRL_NULLPTR ) &&
        ( ptr_threads != SIXTRL_NULLPTR ) &&
        ( NS(KernelConfig_get_arch_id)( kernel_config ) ==
                NS(ARCHITECTURE_CUDA) ) &&
        ( !NS(KernelConfig_needs_update)( kernel_config ) ) )
    {
        NS(Track_particles_line_cuda_debug)<<< *ptr_blocks, *ptr_threads >>>(
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                particles_arg ), pset_index,
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                beam_elem_arg ),
            be_begin_idx, be_end_idx, finish_turn,
            NS(Argument_get_cobjects_buffer_slot_size)( particles_arg ),
            NS(CudaArgument_get_cuda_arg_buffer_as_debugging_register_begin)(
                dbg_register_arg ) );
    }

    return;
}

void NS(BeamMonitor_assign_out_buffer_from_offset_cuda_wrapper)(
    const NS(CudaKernelConfig) *const SIXTRL_RESTRICT kernel_config,
    NS(CudaArgument)* SIXTRL_RESTRICT beam_elem_arg,
    NS(CudaArgument)* SIXTRL_RESTRICT output_arg,
    NS(particle_index_t) const min_turn_id,
    NS(buffer_size_t) const out_buffer_offset_index,
    NS(CudaArgument)* SIXTRL_RESTRICT dbg_register_arg )
{
    dim3 const* ptr_blocks = NS(CudaKernelConfig_get_ptr_const_blocks)(
        kernel_config );

    dim3 const* ptr_threads =
        NS(CudaKernelConfig_get_ptr_const_threads_per_block)( kernel_config );

    if( dbg_register_arg == SIXTRL_NULLPTR )
    {
        SIXTRL_ASSERT( ptr_blocks  != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( ptr_threads != SIXTRL_NULLPTR );

        SIXTRL_ASSERT( !NS(KernelConfig_needs_update)( kernel_config ) );
        SIXTRL_ASSERT( NS(KernelConfig_get_arch_id)( kernel_config ) ==
            NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( output_arg != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( NS(Argument_get_arch_id)( output_arg ) ==
                   NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( output_arg ) );
        SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
            output_arg ) != SIXTRL_NULLPTR );

        SIXTRL_ASSERT( beam_elem_arg != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( NS(Argument_get_arch_id)( beam_elem_arg ) ==
                   NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( beam_elem_arg ) );
        SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
            beam_elem_arg ) != SIXTRL_NULLPTR );

        SIXTRL_ASSERT( min_turn_id >= ( NS(particle_index_t) )0u );
        SIXTRL_ASSERT( out_buffer_offset_index <
            NS(Buffer_get_num_of_objects)(
                NS(Argument_get_const_cobjects_buffer)( output_arg ) ) );

        SIXTRL_ASSERT( NS(Argument_get_cobjects_buffer_slot_size)( output_arg )
            > ( NS(buffer_size_t) )0u );

        NS(BeamMonitor_assign_out_buffer_from_offset_cuda)<<<
            *ptr_blocks, *ptr_threads >>>(
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                beam_elem_arg ),
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                output_arg ), min_turn_id, out_buffer_offset_index,
            NS(Argument_get_cobjects_buffer_slot_size)( output_arg ) );


    }
    else if( ( ptr_blocks != SIXTRL_NULLPTR ) &&
        ( ptr_threads != SIXTRL_NULLPTR ) &&
        ( NS(KernelConfig_get_arch_id)( kernel_config ) ==
                NS(ARCHITECTURE_CUDA) ) &&
        ( !NS(KernelConfig_needs_update)( kernel_config ) ) )
    {
        NS(BeamMonitor_assign_out_buffer_from_offset_cuda_debug)<<<
            *ptr_blocks, *ptr_threads >>>(
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                beam_elem_arg ),
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                output_arg ), min_turn_id, out_buffer_offset_index,
            NS(Argument_get_cobjects_buffer_slot_size)( output_arg ),
            NS(CudaArgument_get_cuda_arg_buffer_as_debugging_register_begin)(
                dbg_register_arg ) );
    }
}

void NS(ElemByElemConfig_assign_out_buffer_from_offset_cuda_wrapper)(
    const NS(CudaKernelConfig) *const SIXTRL_RESTRICT kernel_config,
    NS(CudaArgument)* SIXTRL_RESTRICT config_buffer_arg,
    NS(buffer_size_t) const elem_by_elem_config_index,
    NS(CudaArgument)* SIXTRL_RESTRICT output_arg,
    NS(buffer_size_t) const out_buffer_offset_index,
    NS(CudaArgument)* SIXTRL_RESTRICT dbg_register_arg )
{
    dim3 const* ptr_blocks = NS(CudaKernelConfig_get_ptr_const_blocks)(
        kernel_config );

    dim3 const* ptr_threads =
        NS(CudaKernelConfig_get_ptr_const_threads_per_block)( kernel_config );

    if( dbg_register_arg != SIXTRL_NULLPTR )
    {
        SIXTRL_ASSERT( ptr_blocks  != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( ptr_threads != SIXTRL_NULLPTR );

        SIXTRL_ASSERT( !NS(KernelConfig_needs_update)( kernel_config ) );
        SIXTRL_ASSERT( NS(KernelConfig_get_arch_id)( kernel_config ) ==
            NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( output_arg != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( NS(Argument_get_arch_id)( output_arg ) ==
                   NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( output_arg ) );
        SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
            output_arg ) != SIXTRL_NULLPTR );

        SIXTRL_ASSERT( config_buffer_arg != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( NS(Argument_get_arch_id)( config_buffer_arg ) ==
                   NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( config_buffer_arg ) );
        SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
            config_buffer_arg ) != SIXTRL_NULLPTR );

        SIXTRL_ASSERT(
            NS(Argument_get_cobjects_buffer_slot_size)( config_buffer_arg ) ==
            NS(Argument_get_cobjects_buffer_slot_size)( output_arg ) );

        SIXTRL_ASSERT( out_buffer_offset_index <
            NS(Buffer_get_num_of_objects)(
                NS(Argument_get_const_cobjects_buffer)( output_arg ) ) );

        SIXTRL_ASSERT( NS(Argument_get_cobjects_buffer_slot_size)( output_arg )
            > ( NS(buffer_size_t) )0u );

        NS(ElemByElemConfig_assign_out_buffer_from_offset_cuda)<<<
            *ptr_blocks, *ptr_threads >>>(
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                config_buffer_arg ), elem_by_elem_config_index,
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                output_arg ), out_buffer_offset_index,
            NS(Argument_get_cobjects_buffer_slot_size)( output_arg ) );


    }
    else if( ( ptr_blocks != SIXTRL_NULLPTR ) &&
        ( ptr_threads != SIXTRL_NULLPTR ) &&
        ( NS(KernelConfig_get_arch_id)( kernel_config ) ==
                NS(ARCHITECTURE_CUDA) ) &&
        ( !NS(KernelConfig_needs_update)( kernel_config ) ) )
    {
        NS(ElemByElemConfig_assign_out_buffer_from_offset_cuda_debug)<<<
            *ptr_blocks, *ptr_threads >>>(
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                config_buffer_arg ), elem_by_elem_config_index,
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                output_arg ), out_buffer_offset_index,
            NS(Argument_get_cobjects_buffer_slot_size)( output_arg ),
            NS(CudaArgument_get_cuda_arg_buffer_as_debugging_register_begin)(
                dbg_register_arg ) );
    }
}

void NS(AssignAddressItem_process_managed_buffer_cuda_wrapper)(
    const NS(CudaKernelConfig) *const SIXTRL_RESTRICT kernel_config,
    NS(CudaArgument)* SIXTRL_RESTRICT assign_buffer_arg,
    NS(CudaArgument)* SIXTRL_RESTRICT dest_buffer_arg,
    NS(buffer_size_t) const dest_buffer_id,
    NS(CudaArgument)* SIXTRL_RESTRICT src_buffer_arg,
    NS(buffer_size_t) const src_buffer_id )
{
    dim3 const* ptr_blocks =
        NS(CudaKernelConfig_get_ptr_const_blocks)( kernel_config );

    dim3 const* ptr_threads =
        NS(CudaKernelConfig_get_ptr_const_threads_per_block)( kernel_config );

    NS(buffer_size_t) const assign_slot_size =  NS(Buffer_get_slot_size)(
        NS(Argument_get_const_cobjects_buffer)( dest_buffer_arg ) );

    NS(buffer_size_t) const dest_slot_size = NS(Buffer_get_slot_size)(
        NS(Argument_get_const_cobjects_buffer)( dest_buffer_arg ) );

    NS(buffer_size_t) const src_slot_size = NS(Buffer_get_slot_size)(
        NS(Argument_get_const_cobjects_buffer)( src_buffer_arg ) );

    SIXTRL_ASSERT( ptr_blocks  != SIXTRL_NULLPTR );
    SIXTRL_ASSERT( ptr_threads != SIXTRL_NULLPTR );

    SIXTRL_ASSERT( !NS(KernelConfig_needs_update)( kernel_config ) );
    SIXTRL_ASSERT( NS(KernelConfig_get_arch_id)( kernel_config ) ==
        NS(ARCHITECTURE_CUDA) );

    SIXTRL_ASSERT( assign_buffer_arg != SIXTRL_NULLPTR );
    SIXTRL_ASSERT( NS(Argument_get_arch_id)( assign_buffer_arg ) ==
               NS(ARCHITECTURE_CUDA) );

    SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( assign_buffer_arg ) );
    SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
        assign_buffer_arg ) != SIXTRL_NULLPTR );
    SIXTRL_ASSERT( assign_slot_size > ( NS(buffer_size_t) )0 );

    SIXTRL_ASSERT( dest_buffer_arg != SIXTRL_NULLPTR );
    SIXTRL_ASSERT( NS(Argument_get_arch_id)( dest_buffer_arg ) ==
               NS(ARCHITECTURE_CUDA) );
    SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( dest_buffer_arg ) );
    SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
        dest_buffer_arg ) != SIXTRL_NULLPTR );
    SIXTRL_ASSERT( dest_slot_size > ( NS(buffer_size_t) )0u );
    SIXTRL_ASSERT( dest_buffer_id != (
        NS(buffer_size_t) )SIXTRL_ARCH_ILLEGAL_BUFFER_ID );

    SIXTRL_ASSERT( src_buffer_arg != SIXTRL_NULLPTR );
    SIXTRL_ASSERT( NS(Argument_get_arch_id)( src_buffer_arg ) ==
               NS(ARCHITECTURE_CUDA) );

    SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( src_buffer_arg ) );
    SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
        src_buffer_arg ) != SIXTRL_NULLPTR );
    SIXTRL_ASSERT( src_slot_size > ( NS(buffer_size_t) )0u );
    SIXTRL_ASSERT( src_buffer_id != (
        NS(buffer_size_t) )SIXTRL_ARCH_ILLEGAL_BUFFER_ID );

    NS(AssignAddressItem_process_managed_buffer_cuda)<<< *ptr_blocks,
        *ptr_threads >>>(
        NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
            assign_buffer_arg ), assign_slot_size,
        NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
            dest_buffer_arg ), dest_slot_size, dest_buffer_id,
        NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
            src_buffer_arg ), src_slot_size, src_buffer_id );
}

void NS(Particles_buffer_store_all_addresses_cuda_wrapper)(
    const NS(CudaKernelConfig) *const SIXTRL_RESTRICT kernel_config,
    NS(CudaArgument)* SIXTRL_RESTRICT particles_addresses_arg,
    NS(CudaArgument)* SIXTRL_RESTRICT particles_arg,
    NS(CudaArgument)* SIXTRL_RESTRICT dbg_register_arg )
{
    dim3 const* ptr_blocks = NS(CudaKernelConfig_get_ptr_const_blocks)(
        kernel_config );

    dim3 const* ptr_threads =
        NS(CudaKernelConfig_get_ptr_const_threads_per_block)( kernel_config );

    if( dbg_register_arg == SIXTRL_NULLPTR )
    {
        SIXTRL_ASSERT( ptr_blocks  != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( ptr_threads != SIXTRL_NULLPTR );

        SIXTRL_ASSERT( !NS(KernelConfig_needs_update)( kernel_config ) );
        SIXTRL_ASSERT( NS(KernelConfig_get_arch_id)( kernel_config ) ==
            NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( particles_arg != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( NS(Argument_get_arch_id)( particles_arg ) ==
                   NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)( particles_arg ) );
        SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
            particles_arg ) != SIXTRL_NULLPTR );

        SIXTRL_ASSERT( NS(Argument_get_cobjects_buffer_slot_size)(
            particles_arg ) > ( NS(buffer_size_t) )0u );

        SIXTRL_ASSERT( particles_addresses_arg != SIXTRL_NULLPTR );
        SIXTRL_ASSERT( NS(Argument_get_arch_id)( particles_addresses_arg ) ==
                   NS(ARCHITECTURE_CUDA) );

        SIXTRL_ASSERT( NS(Argument_uses_cobjects_buffer)(
            particles_addresses_arg ) );

        SIXTRL_ASSERT( NS(Argument_get_const_cobjects_buffer)(
            particles_addresses_arg ) != SIXTRL_NULLPTR );

        NS(Particles_buffer_store_all_addresses_cuda)<<<
            *ptr_blocks, *ptr_threads >>>(
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                particles_addresses_arg ),
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                particles_arg ),
            NS(Argument_get_cobjects_buffer_slot_size)( particles_arg ) );


    }
    else if( ( ptr_blocks != SIXTRL_NULLPTR ) &&
        ( ptr_threads != SIXTRL_NULLPTR ) &&
        ( NS(KernelConfig_get_arch_id)( kernel_config ) ==
                NS(ARCHITECTURE_CUDA) ) &&
        ( !NS(KernelConfig_needs_update)( kernel_config ) ) )
    {
        NS(Particles_buffer_store_all_addresses_cuda_debug)<<<
            *ptr_blocks, *ptr_threads >>>(
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                particles_addresses_arg ),
            NS(CudaArgument_get_cuda_arg_buffer_as_cobject_buffer_begin)(
                particles_arg ),
            NS(Argument_get_cobjects_buffer_slot_size)( particles_arg ),
            NS(CudaArgument_get_cuda_arg_buffer_as_debugging_register_begin)(
                dbg_register_arg ) );
    }

    ::hipError_t const err = ::hipDeviceSynchronize();
    SIXTRL_ASSERT( err == ::hipSuccess );

    ( void )err;
}

/* end: sixtracklib/cuda/wrappers/track_job_wrappers.cu */
